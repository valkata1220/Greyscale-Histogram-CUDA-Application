#include "hip/hip_runtime.h"


#include <string.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime_api.h"
#include "stdafx.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
using namespace std;


////////////////////////////////////////////////////////////////////////////////
// GPU routines
////////////////////////////////////////////////////////////////////////////////

__global__ void GrayScale(int *g_outdata, unsigned char *g_indata, int bin_width)
{
	int tid = (threadIdx.x + blockDim.x*blockIdx.x)*3;
	unsigned char gray = (g_indata[tid] + g_indata[tid+1] + g_indata[tid + 2])/3;
	atomicAdd(&g_outdata[gray / bin_width], 1);
	//g_outdata[tid] = gray;
	//g_outdata[tid+1] = gray;
	//g_outdata[tid+2] = gray;
}


int main(int argc, const char **argv)
{
	int num_elements, num_threads, mem_size, num_blocks,bin_count,bin_width;
	int width, height, channels;
	char filename[20];
	char picture[100];
	//bool f;
	unsigned char *din_image;
	unsigned char *dout_image;
	int *out_hist;
	int *histogram;


	cout << "Enter image location:" << endl;
	cin >> picture;

	// User entering blocks count...
	cout << "Enter blocks count:" << endl;
	cin >> bin_count;

	bin_width = 255 / bin_count;
	if (bin_count % 255 != 0) bin_width += 1;

	unsigned char *image = stbi_load(picture, &width, &height, &channels, 3);
	if (!image) cout << "Unsuccessful loading!" << endl;
	else cout << "Image successfuly loaded" << endl;

	findCudaDevice(argc, argv);
	
	histogram = (int*) malloc(bin_count * sizeof(int));
	mem_size = sizeof(char) * (width*height*3);
	checkCudaErrors(hipMalloc((void**)&din_image, mem_size));
	checkCudaErrors(hipMalloc((void**)&out_hist, bin_count * sizeof(int)));
	//checkCudaErrors(hipMalloc((void**)&dout_image, bin_count * sizeof(int)));
	checkCudaErrors(hipMemcpy(din_image, image, mem_size, hipMemcpyHostToDevice));

	
	
	num_elements = width*height;
	num_threads = 1024;
	num_blocks = num_elements/num_threads;

	GrayScale << <num_blocks, num_threads >> > (out_hist, din_image,bin_width);
	getLastCudaError("GrayScale kernel execution failed");

	// copy result from device to host

	//checkCudaErrors(hipMemcpy(image, dout_image, mem_size, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(histogram, out_hist, bin_count * sizeof(int), hipMemcpyDeviceToHost));


	cout << "Enter new filename" << endl;
	cin >> filename;

	//stbi_write_jpg(filename, width, height, channels, image, 100);

	for (int i = 0; i < bin_count; i++)
	{
		printf("Bin � %d - %d\n", i, histogram[i]);
	}
	free(image);

	checkCudaErrors(hipFree(din_image));
	//checkCudaErrors(hipFree(dout_image));
	hipDeviceReset();

	return 0;

}


